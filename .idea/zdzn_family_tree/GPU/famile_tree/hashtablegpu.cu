#include "hip/hip_runtime.h"
/*
 * hashtablegpu.cpp
 *
 *  Created on: 2018年5月17日
 *      Author: 罗锋
 *  基于多GPU的hash表实现
 */
#include ""
#include "hip/hip_runtime.h"
#include "nccl.h"
#include "file_input.h"
#include <src/cudalib.h>
#include <hip/hip_fp16.h>
#include <src/hashtablegpu.h>
#include <iostream>
#include <cmath>
using namespace std;

//递归调用搜索所有的bucketList中index个桶分裂出的所有桶的相关信息更新
//Gd_max为全局最大深度
template<typename T1,typename T2>
void hashtable_gpu<T1,T2>::Recursion_Ld(bucket* bucketList,int i)
 {   int index=i+pow(2,bucketList[i].Ld_init);
     bucketList[index].Ld_init=bucketList[i].Ld_init+1;
     bucketList[index].first_Ld=bucketList[i].Ld_init+1;
     bucketList[index].gpu_index=bucketList[i].gpu_index;
     bucketList[i].Ld_init=bucketList[i].Ld_init+1;
 }

template<typename T1,typename T2>
hashtable_gpu<T1,T2>::hashtable_gpu(int Gd,int Gd_max,int e_key_len,int e_value_len,int GPU_num) {

    (this->extendiblehashtable_N).Gd=Gd;
    (this->extendiblehashtable_N).Gd_max=Gd_max;

    (this->extendiblehashtable_N).bucketNum_init=pow(2,Gd);
    int bucketNum_init=(this->extendiblehashtable_N).bucketNum_init;//初始化为实桶个数
    (this->extendiblehashtable_N).bucketNum_max=pow(2,Gd_max);
    int bucketNum_max=(this->extendiblehashtable_N).bucketNum_max;//最大桶个数

    (this->extendiblehashtable_N).bucketList=(bucket*)malloc((this->extendiblehashtable_N).bucketNum_max*sizeof(bucket));
    bucket* init_bucketList=(this->extendiblehashtable_N).bucketList;

    (this->extendiblehashtable_N).e_key_len=e_key_len;
    (this->extendiblehashtable_N).e_value_len=e_value_len;

//根据Gd转化为二叉树的层次
//  Ld=0                                       node(root)
//  Ld=1                      node(0)                                  node(1)
//  Ld=2          node(00)              node(10)             node(01)             node(11)
//  Ld=3     node(000) node(100)   node(010) node(110) | node(001) node(101)  node(011) node(111)
//  gpu分配   -------------------gpu0-----------------    ---------------------gpu1--------------
//在cpu上初始化为一个深度为Gd_max的满二叉树，并且分配其所属的不同GPU
    int deviceCount=-1;
    if (deviceCount==-1)
	   {CUDACHECK(hipGetDeviceCount(&deviceCount));}

//bucketNum_init分段到每个gpu上,此后从每个属于特定gpu分裂出去的桶都属于该特定gpu
    int sub_len=(int)(bucketNum_init/deviceCount);
    //int yu_len=bucketNum_init%deviceCount;
    printf("sub_len:%d \n",sub_len);

    int* gpu_start_p=(int* )malloc(deviceCount*sizeof(int));
    for(int i=0;i<deviceCount;i++){
    	gpu_start_p[i]=i*sub_len;//属于每个gpu的桶的开始位置
//      printf("gpu_start_p[i]:=%d \n",gpu_start_p[i]);
    }

    //从1号桶开始存储数据-------bucketNum_max号桶
    for(int i=0;i<bucketNum_max;i++)
     {  if(i<bucketNum_init)//初始化所有实桶
        {  init_bucketList[i].Ld=Gd;//初始化时候所有桶的当前深度默认为全部当前深度Gd=3
           init_bucketList[i].first_Ld=Gd;//首次出现的层级深度
           init_bucketList[i].Ld_init=Gd;//初始化时候的层级，只用于初始化分配gpu使用
           init_bucketList[i].is_reliable=1;//实桶

           for(int j=0;j<deviceCount-1;j++){
        	  if(i>=gpu_start_p[j] and i<gpu_start_p[j+1])
				  {init_bucketList[i].gpu_index=j;
	//        	       printf("gpu:=%d,%d \n",init_bucketList[i].gpu_index,i);
				  }
        	  else
        	     { init_bucketList[i].gpu_index=deviceCount-1;
//                   printf("gpu:=%d,%d \n",init_bucketList[i].gpu_index,i);
        	     }
             }
        }
       else{
    	   init_bucketList[i].is_reliable=0;//虚数
       }
     }

//遍历所有已经初始的桶，分解出由这个桶分裂出所有子桶，并赋值同一个gpu序号
//i:=0,gpu_index=0,Ld=3,Ld_init=5,frist=3
//i:=1,gpu_index=0,Ld=3,Ld_init=5,frist=3
    int tmp_i=bucketNum_init;
    while(tmp_i<bucketNum_max)
    { for(int i=0;i<tmp_i;i++)
         Recursion_Ld(init_bucketList,i);
      tmp_i=tmp_i*2;
    }

//按所属gpu将桶分解到不同的device上去
    for(int i=0;i<bucketNum_max;i++){
    	printf("i:=%d,gpu_index=%d,Ld=%d,Ld_init=%d,frist=%d \n",i,init_bucketList[i].gpu_index,init_bucketList[i].Ld,init_bucketList[i].Ld_init,init_bucketList[i].first_Ld);
    }
//       CUDACHECK(hipSetDevice(i));
 }

//分解输入集，提前计算每个桶存储的数据是否超标，是否需要分裂并实现分裂
template<typename T1,typename T2>
void hashtable_gpu<T1,T2>::Predo(char* input,int max_an_len){
    int j=0;
    while(*(input+j*max_an_len)!='&'){
//    	   if(*(result+j*max_an_len)=='\0')
           printf("i=:%d,%s \n",j,input+j*max_an_len);
           j++;
     }
     cout<<"hehe:"<<j<<endl;
}

template<typename T1,typename T2>
hashtable_gpu<T1,T2>::~hashtable_gpu() {
	// TODO Auto-generated destructor stub
}

template class hashtable_gpu<byte,byte>;
