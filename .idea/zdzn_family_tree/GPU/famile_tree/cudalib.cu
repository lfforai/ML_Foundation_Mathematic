#include "hip/hip_runtime.h"
/*
 * cudalib.cpp
 *
 *  Created on: 2018年5月17日
 *      Author: root
 */

//#include <src/cudalib.h>
//#include <stdio.h>
//#include "hip/hip_runtime.h"
//#include "nccl.h"
//#include <string>
//#include "src/cudalib.h"
#include ""
#include "hip/hip_runtime.h"
#include "nccl.h"
#include "file_input.h"
#include <src/cudalib.h>
#include <hip/hip_fp16.h>

#include <string>
using namespace std;

//统计最大祖先个数、最大祖先长度和最大单条长度
template <class T>
__host__ __device__ void len(const char*info,T*result)
{
  int i=0;//index
  char frist_mark=(byte)0;
  while(*(info+i)!='@'){
	  if((*(info+i)=='.' or *(info+i)=='_')){
		  result[0]=(T)((int)result[0]+(int)1);
	  }

	  if(frist_mark==(byte)1)//if frist '.'
	  {result[1]=(T)((int)result[1]+(int)1);}

	  if(frist_mark==(byte)0 and *(info+i)=='.')//if frist '.'
	  { frist_mark=(byte)1;
	  }
	  i=i-1;
  }
  result[2]=(T)(abs(i)-2);
}

template <class T>
__global__ void split_global(T* dum, char* info,long start,long length,int dimblock)
{       extern __shared__ byte s[];
        if (threadIdx.x==0){
           memset(s,(byte)0,3*dimblock*sizeof(T));
        }
    	__syncthreads();

//		T* temp=(T*)malloc(2*sizeof(T));
		T temp[3];
		long length_N = length;
		int step = gridDim.x*blockDim.x;
		const long start_P=start;//开始的位置
		long start_N =threadIdx.x+blockIdx.x*blockDim.x;
		for(long start=start_N;start<length_N;start=start+step)
		   {
			  if((char)*(info+start+start_P)=='\n')
		        {  temp[0]=0;
				   temp[1]=0;
				   temp[2]=0;
		           len(info+start+start_P,temp);
		           if((int)temp[0]>(int)s[threadIdx.x*3]){
		        	   s[threadIdx.x*3]=temp[0];
		           }

		           if((int)temp[1]>(int)s[threadIdx.x*3+1]){
		        	   s[threadIdx.x*3+1]=temp[1];
		           }

		           if((int)temp[2]>(int)s[threadIdx.x*3+2]){
		        	   s[threadIdx.x*3+2]=temp[2];
		           }

		        }
		    }
		free(temp);

		//同步
		__syncthreads();
		if(threadIdx.x==0)
		memcpy(dum+3*blockIdx.x*blockDim.x*sizeof(T),s,3*blockDim.x*sizeof(T));
		__syncthreads();
}

//切割出所有祖先，为放入hash表用
//dimGrid_N, dimBlock_N,0,s[i]>>>(d_result[i]+h_len_result[deviceCount-2]*max_an_len*max_an_num,d_info[i],max_an_len,max_an_num,(deviceCount-1)*sub_length,sub_length+yu,dimBlock_N
template <class T>
__global__ void scut2ancestors(char* des,int max_an_len,int max_an_num,char* info,long start,long length,unsigned long long int* mark,int dimblock)
{       //为每个thread分配空间记录当前分解记录的各"_"和“.”的位置
	    extern __shared__ byte s[];
		long length_N = length;
		int step = gridDim.x*blockDim.x;
		const long start_P=start;//开始的位置
		long start_N =threadIdx.x+blockIdx.x*blockDim.x;
		if(threadIdx.x==0){
			*mark=0;
			memset(s,(byte)0,max_an_num*dimblock*sizeof(T));
		}
		__syncthreads();

		for(long start=start_N;start<length_N;start=start+step)
		   {
			  if((char)*(info+start+start_P)=='\n')
		        { //所有的相对位置都是相对于每个‘\n’回车符号来的
				  //1、-----------------------------------------------------
				  int dian_i=0;//第一个"."前面的最大祖先的根位置
				  int last_i=0;//开始的"@"

				  int first_dian_mark=0;//是否第一个"."
				  int i=0;
				  while(*(info+start+start_P+i)!='@')
		          {if(*(info+start+start_P+i)=='.' and first_dian_mark==0)
		        	  {dian_i=i;
		        	   first_dian_mark=1;
		        	  }
		           i=i-1;
		          }
				  last_i=i;
//				  printf("a:=%c \n",*(info+start+start_P+i));
				  dian_i=dian_i-1;//最大祖先的开始位置info+start+start_P+dian_i
				  last_i=last_i+1;//最大祖先的结束位置info+start+start_P+last_i
				  //----------------------------------------------------


//				  //2、记录各级祖先节点的位置----"."和“—”
				  int last_i_N=last_i;//保留祖先开始位置记录
				  int dian_i_N=dian_i+1;//“.”位置
				  int an_num=0;//祖先数目
				  while(last_i<=dian_i_N and an_num<max_an_num){
					  if(*(info+start+start_P+last_i)=='.' or *(info+start+start_P+last_i)=='_'){
						s[threadIdx.x*max_an_num+an_num]=last_i;
//						printf("*(info+start+start_P+s[w]):%c,theid:=%d \n",*(info+start+start_P+s[threadIdx.x*max_an_num+an_num]),threadIdx.x*max_an_num+an_num);
						an_num=an_num+1;
					  }
					  last_i=last_i+1;
				  }

//				  //3、依次输出各祖先节点
				  an_num=0;
				  unsigned long long int  position=0;
				  //(*(info+start+start_P+s[threadIdx.x*max_an_num+an_num])=='.' or *(info+start+start_P+s[threadIdx.x*max_an_num+an_num])=='_')  and
				  while((*(info+start+start_P+s[threadIdx.x*max_an_num+an_num])=='.' or *(info+start+start_P+s[threadIdx.x*max_an_num+an_num])=='_') and an_num<max_an_num){
					   position=(unsigned long long int )atomicAdd((unsigned long long int  *)mark,(unsigned long long int )1);
					   memcpy(des+position*max_an_len,info+start+start_P+last_i_N,s[threadIdx.x*max_an_num+an_num]-last_i_N);
					   *(des+position*max_an_len+s[threadIdx.x*max_an_num+an_num]-last_i_N)='\0';
//					   if(*(des+position*max_an_len+s[threadIdx.x*max_an_num+an_num]-last_i_N-1)=='.')
//					   {char* temp=(char *)malloc(-last_i_N);
//					    memcpy(temp,info+start+start_P+last_i_N,-last_i_N);
//					    *(temp-last_i_N)='\0';
//						printf("n:=%s||%s||%c,%c,%d,%d \n",des+position*max_an_len,temp,*(info+start+start_P+dian_i),*(info+start+start_P+last_i_N),(int)s[threadIdx.x*max_an_num+an_num],an_num);
//					   }
					   an_num=an_num+1;
				  }

				  //放入整条记录
				  position=(unsigned long long int )atomicAdd((unsigned long long int  *)mark,(unsigned long long int )1);
				  memcpy(des+position*max_an_len,info+start+start_P+last_i_N,-last_i_N+1);
				  *(des+position*max_an_len-last_i_N)='\0';
//				  if(*(des+position*max_an_len)=='\0')
//				  {char* temp=(char *)malloc(-last_i_N+1);
//				   memcpy(temp,info+start+start_P+last_i_N,-last_i_N);
//				   *(temp-last_i_N)='\0';
//				   printf("ok::%s||%s||%c,%c,%d,%d \n",des+position*max_an_len,temp,*(info+start+start_P+dian_i),*(info+start+start_P+last_i_N),(int)s[threadIdx.x*max_an_num+an_num],an_num);
//				   delete temp;
//				  }
//				  printf("ok:=%s \n",des+position*max_an_len);

				  __syncthreads();
				  if(threadIdx.x==0){memset(s,(byte)0,max_an_num*dimblock*sizeof(T));}
				  __syncthreads();
		        }
		    }
}

template __host__ __device__ void len<ubyte>(const char*,ubyte *);
template __host__ __device__ void len<byte>(const char*,byte *);
template __host__ __device__ void len<int>(const char*,int *);

template __global__ void split_global<ubyte>(ubyte*, char*,long,long,int);
template __global__ void split_global<byte>(byte*, char*,long,long,int);
template __global__ void split_global<int>(int*, char*,long,long,int);

template __global__ void scut2ancestors<byte>(char*,int ,int ,char*,long,long,unsigned long long int *,int);
template __global__ void scut2ancestors<ubyte>(char*,int ,int ,char*,long,long,unsigned long long int *,int);
template __global__ void scut2ancestors<int>(char*,int ,int ,char*,long,long,unsigned long long int *,int);
