#include "hip/hip_runtime.h"

/**
 * 作用：求所有key的最大祖先个数和最大key长度
 * 作者：罗峰
 */
#include "nccl.h"
#include ""
#include "hip/hip_runtime.h"
#include <src/cudalib.h>
#include <src/split.h>

template<typename T>
split<T>::split() {
	// TODO Auto-generated constructor stub
}

template<typename T>
split<T>::~split() {
	// TODO Auto-generated destructor stub
}

//返回所有key值中分解出的最多的一个祖先个数
//例如：MDNY.SZWQ.11MD#1.MlyGnCptLo,祖先个数为3
//返回keys列表中单个key最大祖先个数、最大祖先字符串长度，本身字符串最大长度
template<typename T>
T* split<T>::max_ancestors_num(file_input::info* info_of_key,int GPU_num){
	int dimGrid_N=224;
	int dimBlock_N=256;
	int result_n=3;//需要输出几种结果：目前为3

    T* max=(T*)malloc(result_n*sizeof(T));//max[0]:祖先最大数，max[1]:key的最大长度
    max[0]=(T)0;
    max[1]=(T)0;
    max[2]=(T)0;

    char* keys_data=info_of_key->data;
//  long row_num=info_of_key->total_row;
    long buffer_size=info_of_key->total_size;//字节数
    int deviceCount=GPU_num;

    if (GPU_num==-1)
	   {CUDACHECK(hipGetDeviceCount(&deviceCount));}

    //#记录每个key的最大祖先个数和最大长度的数组
    T** h_num=(T **)malloc(deviceCount*sizeof(T*));
    T** d_num=(T **)malloc(deviceCount*sizeof(T*));
    char** d_info=(char **)malloc(deviceCount*sizeof(char*));

    long yu= buffer_size%deviceCount;
    long sub_length=buffer_size/deviceCount;

    for(int i=0;i<2;i++)
    {hipOccupancyMaxPotentialBlockSize(
		&dimGrid_N,
		&dimBlock_N,
		(void*)split_global<T>,
		result_n*dimBlock_N*sizeof(T),
		2048);
        printf("第%d次：dimGrid_N=:%d,dimBlock_N:=%d \n",i,dimGrid_N,dimBlock_N);
    }

    //此处假设所有的祖先不超过255个，所以采用char
    for(int i=0;i<deviceCount;i++){
    	CUDACHECK(hipSetDevice(i));
    	h_num[i]=(T *)malloc(result_n*dimGrid_N*dimBlock_N*sizeof(T));
        CUDACHECK(hipMalloc(d_num+i,result_n*dimGrid_N*dimBlock_N*sizeof(T)));
        CUDACHECK(hipMalloc(d_info+i,buffer_size* sizeof(char)));
        if (i==0){//gpu较多情况下使用nccl，在gpu较少情况下可以不使用nccl，这里统一使用nccl无论gpu个数
           CUDACHECK(hipMemcpy(d_info[i],keys_data,buffer_size*sizeof(char), hipMemcpyHostToDevice));
        }
    }

//把 d_info[0]通过nccl的boadcast到d_info[i]上去--------------------开始
    if(deviceCount>0)
     {hipStream_t* s = (hipStream_t*)malloc(sizeof(hipStream_t)*deviceCount);
     ncclComm_t* comms=(ncclComm_t*)malloc(deviceCount*sizeof(ncclComm_t));
     //managing deviceCount devices
     int* devs=(int *)malloc(deviceCount*sizeof(int));
     for(int i=0;i<deviceCount;i++){
    	  CUDACHECK(hipSetDevice(i));
    	  devs[i]=i;
    	  CUDACHECK(hipStreamCreate(s+i));
     }
     //initializing NCCL
     NCCLCHECK(ncclCommInitAll(comms,deviceCount, devs));
     //calling NCCL communication API. Group API is required when using
     //multiple devices per thread
     NCCLCHECK(ncclGroupStart());
     for (int i = 0; i < deviceCount; ++i)
   	     NCCLCHECK(ncclBcast((void*)d_info[i],buffer_size,ncclChar,0,comms[i], s[i]));
     NCCLCHECK(ncclGroupEnd());
     //synchronizing on CUDA streams to wait for completion of NCCL operation
     for (int i = 0; i < deviceCount; ++i) {
       CUDACHECK(hipSetDevice(i));
       CUDACHECK(hipStreamSynchronize(s[i]));
     }
     //finalizing NCCL
      for(int i = 0; i <deviceCount; ++i)
          {ncclCommDestroy(comms[i]);
           CUDACHECK(hipStreamDestroy(s[i]));}
     }
//把 d_info[0]通过nccl的boadcast到d_info[i]上去--------------------开始

      hipStream_t* s = (hipStream_t*)malloc(sizeof(hipStream_t)*deviceCount);
      for(int i=0;i<deviceCount;i++){
     	  CUDACHECK(hipSetDevice(i));
     	  CUDACHECK(hipStreamCreate(s+i));
      }

      for(int i=0;i<deviceCount;i++)
      {   CUDACHECK(hipSetDevice(i));
          if (i!=deviceCount-1)
          {
    	   split_global<T><<<dimGrid_N, dimBlock_N,result_n*dimBlock_N*sizeof(T),s[i]>>>(d_num[i],d_info[i],i*sub_length,sub_length,dimBlock_N);
    	   //2*dimBlock_N*sizeof(T)
          }
          else
          {
    	   split_global<T><<<dimGrid_N, dimBlock_N,result_n*dimBlock_N*sizeof(T),s[i]>>>(d_num[i],d_info[i],(deviceCount-1)*sub_length,sub_length+yu,dimBlock_N);
          }
      }

      for (int i = 0;i < deviceCount;i++)
        {
            CUDACHECK(hipSetDevice(i));
            CUDACHECK(hipMemcpyAsync(h_num[i],d_num[i],result_n*dimGrid_N*dimBlock_N*sizeof(T),hipMemcpyDeviceToHost,s[i]));
        }


      for (int i = 0;i < deviceCount;i++)
      {
          CUDACHECK(hipSetDevice(i));
          CUDACHECK(hipStreamSynchronize(s[i]));
      }

      for(int i = 0; i <deviceCount; ++i)
           {CUDACHECK(hipStreamDestroy(s[i]));}

      for (int i = 0; i < deviceCount;i++)
      {  for (int j = 0; j < dimGrid_N*dimBlock_N;j++)
          {
        	 if((int)h_num[i][j*result_n]>(int)max[0]){
        		 max[0]=h_num[i][j*result_n];
//        		 printf("max[0]：=%d \n",max[0]);
        	 }

        	 if((int)h_num[i][j*result_n+1]>(int)max[1]){
        		 max[1]=h_num[i][j*result_n+1];
//        		 printf("max[1]：=%d \n",max[1]);
        	 }

        	 if((int)h_num[i][j*result_n+2]>(int)max[2]){
        		 max[2]=h_num[i][j*result_n+2];
//        		 printf("max[1]：=%d \n",max[1]);
        	 }
//        	     if(j<2 and (h_num[i][j*2]!=0 or h_num[i][j*2+1]!=0))
//        	     printf("max[0]=%d,max[1]=%d,i:=%d,j:=%d \n",(int)h_num[i][j*2],(int)h_num[i][j*2+1],i,j);
          }
      }
      max[2]=max[2]+1;//程序有问题统计出来最大值少1？
      max[1]=max[1]+1;//
      printf("%d,%d,%d \n",max[0],max[1],max[2]);

      for (int i = 0;i < deviceCount;i++)
      {
//        CUDACHECK(hipSetDevice(i));
          CUDACHECK(hipFree(d_info[i]));
          CUDACHECK(hipFree(d_num[i]));
      }
      return max;
};


//int num记录条数，max_an_len是祖先长度,max_an_num祖先个数，
//按6个祖先，每个32最大长度，有1000万个点计算需要消耗内存为1.8g存储
template<typename T>
char* split<T>::cut2ancestors(file_input::info* info_of_key,int max_an_num,int max_an_len,int GPU_num){
    int num=info_of_key->total_row;
    char* keys_data=info_of_key->data;
    long buffer_size=info_of_key->total_size;//字节数
	int dimGrid_N=224;
	int dimBlock_N=256;

	int deviceCount=2;
	if (GPU_num==-1)
	   {CUDACHECK(hipGetDeviceCount(&deviceCount));}

	//每个GPU分解一部分
	char** h_result=(char**)malloc(deviceCount*sizeof(char*));
    char** d_result=(char **)malloc(deviceCount*sizeof(char*));
    char** d_info=(char **)malloc(deviceCount*sizeof(char*));

    //对输入数据的切割
    long yu= buffer_size%deviceCount;
    long sub_length=buffer_size/deviceCount;

    //对应输入数据对输出数据进行切割------------------------------
    //存放每个gpu开始的位置索引
    int* h_len_result=(int*)malloc(deviceCount*sizeof(int));
    for(int i=0;i<deviceCount;i++)
    {  if(i!=0)
       {for(int j=0;j<num-2;j++)
          {if(i*sub_length>=info_of_key->split_mark[j] and i*sub_length<info_of_key->split_mark[j+1])
        	  if(info_of_key->split_mark[info_of_key->split_mark[j]]!='\n')
        	     h_len_result[i]=j+1;
        	  else
        		 h_len_result[i]=j;
    	   }
       }
       else
       {h_len_result[0]=0;}
       printf(" h_len_result[i]:=%d \n",h_len_result[i]);
    }

    //-------------------------切割结束----------------------------

    for(int i=0;i<2;i++)
    {hipOccupancyMaxPotentialBlockSize(
		&dimGrid_N,
		&dimBlock_N,
		(void*)scut2ancestors<T>,
		max_an_num*dimBlock_N*sizeof(T),
		2048);
        printf("第%d次：dimGrid_N=:%d,dimBlock_N:=%d \n",i,dimGrid_N,dimBlock_N);
    }

    //初始化所有gpu上数据
    for(int i=0;i<deviceCount;i++){
    	CUDACHECK(hipSetDevice(i));
    	h_result[i]=(char *)malloc(num*max_an_len*max_an_num*sizeof(char));
        CUDACHECK(hipMalloc(d_result+i,num*max_an_len*max_an_num*sizeof(char)));
        CUDACHECK(hipMalloc(d_info+i,buffer_size* sizeof(char)));
        if (i==0){//gpu较多情况下使用nccl，在gpu较少情况下可以不使用nccl，这里统一使用nccl无论gpu个数
           CUDACHECK(hipMemcpy(d_info[i],keys_data,buffer_size*sizeof(char), hipMemcpyHostToDevice));
        }
    }

//把 d_info[0]通过nccl的boadcast到d_info[i]上去--------------------开始
    if(deviceCount>0)
    { hipStream_t* s = (hipStream_t*)malloc(sizeof(hipStream_t)*deviceCount);
     ncclComm_t* comms=(ncclComm_t*)malloc(deviceCount*sizeof(ncclComm_t));
     //managing deviceCount devices
     int* devs=(int *)malloc(deviceCount*sizeof(int));
     for(int i=0;i<deviceCount;i++){
    	  CUDACHECK(hipSetDevice(i));
    	  devs[i]=i;
    	  CUDACHECK(hipStreamCreate(s+i));
     }

     //initializing NCCL
     NCCLCHECK(ncclCommInitAll(comms,deviceCount, devs));
     //calling NCCL communication API. Group API is required when using
     //multiple devices per thread
     NCCLCHECK(ncclGroupStart());
     for (int i = 0; i < deviceCount; ++i)
   	     NCCLCHECK(ncclBcast((void*)d_info[i],buffer_size,ncclChar,0,comms[i], s[i]));
     NCCLCHECK(ncclGroupEnd());
     //synchronizing on CUDA streams to wait for completion of NCCL operation
     for (int i = 0; i < deviceCount; ++i) {
       CUDACHECK(hipSetDevice(i));
       CUDACHECK(hipStreamSynchronize(s[i]));
     }
     //finalizing NCCL
     for(int i = 0; i <deviceCount; ++i)
          {ncclCommDestroy(comms[i]);
           CUDACHECK(hipStreamDestroy(s[i]));}
    }
//把 d_info[0]通过nccl的boadcast到d_info[i]上去--------------------开始

        hipStream_t*  s = (hipStream_t*)malloc(sizeof(hipStream_t)*deviceCount);
        for(int i=0;i<deviceCount;i++){
       	  CUDACHECK(hipSetDevice(i));
       	  CUDACHECK(hipStreamCreate(s+i));
        }

        //记录当前记录存放位置的变量
        unsigned long long int ** p_mark=(unsigned long long int ** )malloc(deviceCount*sizeof(unsigned long long int *));
        for(int i=0;i<deviceCount;i++){
           CUDACHECK(hipSetDevice(i));
           CUDACHECK(hipMalloc(p_mark+i,sizeof(unsigned long long int)));
        }


        for(int i=0;i<deviceCount;i++)
        {   CUDACHECK(hipSetDevice(i));
            if (i==0)
            {
            scut2ancestors<T><<<dimGrid_N, dimBlock_N,max_an_num*dimBlock_N*sizeof(T),s[i]>>>(d_result[i],max_an_len,max_an_num,d_info[i],i*sub_length,sub_length,p_mark[i],dimBlock_N);
            }
            else
            { if(i==deviceCount-1)
              scut2ancestors<T><<<dimGrid_N, dimBlock_N,max_an_num*dimBlock_N*sizeof(T),s[i]>>>(d_result[i]+h_len_result[i]*max_an_len*max_an_num,max_an_len,max_an_num,d_info[i],i*sub_length,sub_length+yu,p_mark[i],dimBlock_N);
              else
              scut2ancestors<T><<<dimGrid_N, dimBlock_N,max_an_num*dimBlock_N*sizeof(T),s[i]>>>(d_result[i]+h_len_result[i]*max_an_len*max_an_num,max_an_len,max_an_num,d_info[i],i*sub_length,sub_length,p_mark[i],dimBlock_N);
            }
        }


        for (int i = 0;i < deviceCount;i++)
          {
              CUDACHECK(hipSetDevice(i));
              CUDACHECK(hipMemcpyAsync(h_result[i],d_result[i],num*max_an_len*max_an_num*sizeof(char),hipMemcpyDeviceToHost,s[i]));
          }


        unsigned long long int** h_mark=(unsigned long long int**)malloc(sizeof(unsigned long long int *));
        for(int i=0;i<deviceCount;i++){
                h_mark[i]=(unsigned long long int*)malloc(sizeof(unsigned long long int));
         }
        for (int i = 0;i < deviceCount;i++)
             {
                 CUDACHECK(hipSetDevice(i));
                 CUDACHECK(hipMemcpyAsync(h_mark[i],p_mark[i],sizeof(unsigned long long int), hipMemcpyDeviceToHost,s[i]));
             }

        for (int i = 0;i < deviceCount;i++)
        {
            CUDACHECK(hipSetDevice(i));
            CUDACHECK(hipStreamSynchronize(s[i]));
        }

        unsigned long long int total=0;
        for(int i = 0; i <deviceCount;i++)
             {CUDACHECK(hipStreamDestroy(s[i]));
              total=total+*h_mark[i];
              printf("%dl  \n",*h_mark[i]);
             }
        printf("%dl  \n",total);
        char* result=(char *)malloc(total*max_an_len*sizeof(char));

        total=0;
        for(int i = 0; i<deviceCount;i++)
         {  memcpy(result+total*max_an_len,h_result[i]+h_len_result[i]*max_an_len*max_an_num,*h_mark[i]*max_an_len*sizeof(char));
        	total=total+*h_mark[i];
         }

       memset(result+total*max_an_len,'&',max_an_len);

       for (int i = 0;i < deviceCount;i++)
       {
           CUDACHECK(hipFree(d_info[i]));
           CUDACHECK(hipFree(d_result[i]));
           CUDACHECK(hipFree(p_mark[i]));
       }

//       int j=0;
//       while(*(result+j*max_an_len)!='&'){
////    	   if(*(result+j*max_an_len)=='\0')
//              printf("i=:%d,%s \n",j,result+j*max_an_len);
//              j++;
//        }
//       printf("i=:%d,%c \n",j,(char)*(result+j*max_an_len+1));
//       printf("max:=%d",max_an_len);
       return result;
}

template class split<byte>;
template class split<ubyte>;
template class split<int>;

