#include "hip/hip_runtime.h"
/*
 * hashtablegpu.cpp
 *
 *  Created on: 2018年5月17日
 *      Author: 罗锋
 *  基于多GPU的hash表实现
 */
#include ""
#include "hip/hip_runtime.h"
#include "nccl.h"
#include "file_input.h"
#include <src/cudalib.h>
#include <hip/hip_fp16.h>
#include <src/hashtablegpu.h>
#include <cmath>

//递归调用搜索所有的bucketList中index个桶分裂出的所有桶的相关信息更新
//Gd_max为全局最大深度
template<typename T1,typename T2>
void hashtable_gpu<T1,T2>::Recursion_Ld(bucket* bucketList,int i)
 {   int index=i+pow(2,bucketList[i].Ld_init);
     bucketList[index].Ld_init=bucketList[i].Ld_init+1;
     bucketList[index].first_Ld=bucketList[i].Ld_init+1;
     bucketList[index].gpu_index=bucketList[i].gpu_index;
     bucketList[i].Ld_init=bucketList[i].Ld_init+1;
 }

template<typename T1,typename T2>
hashtable_gpu<T1,T2>::hashtable_gpu(int Gd,int Gd_max,int e_key_len,int e_value_len,int GPU_num) {

    (this->extendiblehashtable_N).Gd=Gd;
    (this->extendiblehashtable_N).Gd_max=Gd_max;


    (this->extendiblehashtable_N).bucketNum_init=pow(2,Gd);
    int bucketNum_init=(this->extendiblehashtable_N).bucketNum_init;
    (this->extendiblehashtable_N).bucketNum_max=pow(2,Gd_max);
    int bucketNum_max=(this->extendiblehashtable_N).bucketNum_max;

    (this->extendiblehashtable_N).bucketList=(bucket*)malloc((this->extendiblehashtable_N).bucketNum_max*sizeof(bucket));
    bucket* init_bucketList=(this->extendiblehashtable_N).bucketList;

    (this->extendiblehashtable_N).e_key_len=e_key_len;
    (this->extendiblehashtable_N).e_value_len=e_value_len;

//根据Gd转化为二叉树的层次
//  Ld=0                                       node(root)
//  Ld=1                      node(0)                                  node(1)
//  Ld=2          node(00)              node(10)             node(01)             node(11)
//  Ld=3     node(000) node(100)   node(010) node(110) | node(001) node(101)  node(011) node(111)
//  gpu分配   -------------------gpu0-----------------    ---------------------gpu1--------------
//在cpu上初始化为一个深度为Gd_max的满二叉树，并且分配其所属的不同GPU
    int deviceCount=-1;
    if (deviceCount==-1)
	   {CUDACHECK(hipGetDeviceCount(&deviceCount));}

//bucketNum_init分段到每个gpu上,此后从每个属于特定gpu分裂出去的桶都属于该特定gpu
    int sub_len=(int)(bucketNum_init/deviceCount);
    //int yu_len=bucketNum_init%deviceCount;
    printf("sub_len:%d \n",sub_len);

    int* cpu_start_p=(int* )malloc(deviceCount*sizeof(int));
    for(int i=0;i<deviceCount;i++){
    	cpu_start_p[i]=i*sub_len;//属于每个gpu的桶的开始位置
//      printf("cpu_start_p[i]:=%d \n",cpu_start_p[i]);
    }

    //从1号桶开始存储数据-------bucketNum_max号桶
    for(int i=0;i<bucketNum_max;i++)
     {  if(i<bucketNum_init)
        {  init_bucketList[i].Ld=Gd;//初始化时候所有桶都在全局深度默认为Gd=3
           init_bucketList[i].first_Ld=Gd;
           init_bucketList[i].Ld_init=Gd;
           init_bucketList[i].is_reliable=1;

           for(int j=0;j<deviceCount-1;j++){
        	  if(i>=cpu_start_p[j] and i<cpu_start_p[j+1])
        		  {init_bucketList[i].gpu_index=j;
//        	       printf("gpu:=%d,%d \n",init_bucketList[i].gpu_index,i);
        		  }
        	  else
        	     { init_bucketList[i].gpu_index=deviceCount-1;
//                   printf("gpu:=%d,%d \n",init_bucketList[i].gpu_index,i);
        	     }
             }
        }
     }

//遍历所有已经初始的桶，分解出由这个桶分裂出所有子桶，并赋值同一个gpu序号
//i:=0,gpu_index=0,Ld=3,Ld_init=5,frist=3
//i:=1,gpu_index=0,Ld=3,Ld_init=5,frist=3
    int tmp_i=bucketNum_init;
    while(tmp_i<bucketNum_max)
    { for(int i=0;i<tmp_i;i++)
         Recursion_Ld(init_bucketList,i);
      tmp_i=tmp_i*2;
    }

//按所属gpu将桶分解到不同的device上去

//    for(int i=0;i<bucketNum_max;i++){
//    	printf("i:=%d,gpu_index=%d,Ld=%d,Ld_init=%d,frist=%d \n",i,init_bucketList[i].gpu_index,init_bucketList[i].Ld,init_bucketList[i].Ld_init,init_bucketList[i].first_Ld);
//    }

//       CUDACHECK(hipSetDevice(i));
 }

//分解输入集，提前计算每个桶存储的数据是否超标，是否需要分裂并实现分裂
template<typename T1,typename T2>
void hashtable_gpu<T1,T2>::Predo(){

}


template<typename T1,typename T2>
hashtable_gpu<T1,T2>::~hashtable_gpu() {
	// TODO Auto-generated destructor stub
}

template class hashtable_gpu<byte,byte>;
