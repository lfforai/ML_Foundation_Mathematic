#include "hip/hip_runtime.h"
/*
 * cudalib.cpp
 *
 *  Created on: 2018年5月17日
 *      Author: root
 */

//#include <src/cudalib.h>
//#include <stdio.h>
//#include "hip/hip_runtime.h"
//#include "nccl.h"
//#include <string>
//#include "src/cudalib.h"
#include ""
#include "hip/hip_runtime.h"
#include "nccl.h"
#include "file_input.h"
#include <src/cudalib.h>
#include <hip/hip_fp16.h>

#include <string>
using namespace std;

//统计最大祖先个数、最大祖先长度和最大单条长度
template <class T>
__host__ __device__ void len(const char*info,T*result)
{
  int i=0;//index
  char frist_mark=(byte)0;
  while(*(info+i)!='@'){
	  if((*(info+i)=='.' or *(info+i)=='_')){
		  result[0]=(T)((int)result[0]+(int)1);
	  }

	  if(frist_mark==(byte)1)//if frist '.'
	  {result[1]=(T)((int)result[1]+(int)1);}

	  if(frist_mark==(byte)0 and *(info+i)=='.')//if frist '.'
	  { frist_mark=(byte)1;
	  }
	  i=i-1;
  }
  result[2]=(T)(abs(i)-2);
}

template <class T>
__global__ void split_global(T* dum, char* info,long start,long length,int dimblock)
{       extern __shared__ byte s[];
        if (threadIdx.x==0){
           memset(s,(byte)0,3*dimblock*sizeof(T));
        }
    	__syncthreads();
//		T* temp=(T*)malloc(2*sizeof(T));
		T temp[3];
		long length_N = length;
		int step = gridDim.x*blockDim.x;
		const long start_P=start;//开始的位置
		long start_N =threadIdx.x+blockIdx.x*blockDim.x;
		for(long start=start_N;start<length_N;start=start+step)
		   {
			  if((char)*(info+start+start_P)=='\n')
		        {  temp[0]=0;
				   temp[1]=0;
		           len(info+start+start_P,temp);
		           if((int)temp[0]>(int)s[threadIdx.x*3]){
		        	   s[threadIdx.x*3]=temp[0];
		           }


		           if((int)temp[1]>(int)s[threadIdx.x*3+2]){
		        	   s[threadIdx.x*3+1]=temp[1];
		           }

		           if((int)temp[2]>(int)s[threadIdx.x*3+2]){
		        	   s[threadIdx.x*3+2]=temp[2];
		           }

		        }
		    }
		delete temp;

		//同步
		__syncthreads();
		if(threadIdx.x==0)
		memcpy(dum+3*blockIdx.x*blockDim.x*sizeof(T),s,3*blockDim.x*sizeof(T));
		__syncthreads();
}

//切割出所有祖先，为放入hash表用
//dimGrid_N, dimBlock_N,0,s[i]>>>(d_result[i]+h_len_result[deviceCount-2]*max_an_len*max_an_num,d_info[i],max_an_len,max_an_num,(deviceCount-1)*sub_length,sub_length+yu,dimBlock_N
template <class T>
__global__ void scut2ancestors(char* des,long max_an_len,char* info,long start,long length)
{
		long length_N = length;
		int step = gridDim.x*blockDim.x;
		const long start_P=start;//开始的位置
		long start_N =threadIdx.x+blockIdx.x*blockDim.x;
		for(long start=start_N;start<length_N;start=start+step)
		   {
			  if((char)*(info+start+start_P)=='\n')
		        {

		        }
		    }
		delete temp;

		//同步
		__syncthreads();
		if(threadIdx.x==0)
		memcpy(dum+3*blockIdx.x*blockDim.x*sizeof(T),s,3*blockDim.x*sizeof(T));
		__syncthreads();
}



template __host__ __device__ void len<ubyte>(const char*,ubyte *);
template __host__ __device__ void len<byte>(const char*,byte *);
template __global__ void split_global<ubyte>(ubyte*, char*,long,long,int);
template __global__ void split_global<byte>(byte*, char*,long,long,int);

template __host__ __device__ void len<int>(const char*,int *);
template __global__ void split_global<int>(int*, char*,long,long,int);


