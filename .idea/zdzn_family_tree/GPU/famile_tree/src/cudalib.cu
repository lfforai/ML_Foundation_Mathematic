#include "hip/hip_runtime.h"
/*
 * cudalib.cpp
 *
 *  Created on: 2018年5月17日
 *      Author: root
 */

//#include <src/cudalib.h>
//#include <stdio.h>
//#include "hip/hip_runtime.h"
//#include "nccl.h"
//#include <string>
//#include "src/cudalib.h"
#include ""
#include "hip/hip_runtime.h"
#include "nccl.h"
#include "file_input.h"
#include <src/cudalib.h>
#include <hip/hip_fp16.h>

#include <string>
using namespace std;

template <class T>
__host__ __device__ void len(const char*info,T*result)
{
  int i=0;//index
  char frist_mark=0;
  while(*(info+i)!='@'){

	  if((*(info+i)=='.' or *(info+i)=='_')){
		  result[0]=result[0]+1;
	  }

	  if(frist_mark==1)//if frist '.'
	  {result[1]=result[1]+1;}

	  if(frist_mark==0 and *(info+i)=='.')//if frist '.'
	  { frist_mark=1;
	  }
	  i=i-1;
  }
}

template <class T>
__global__ void split_global(T* dum, char* info,long start,long length)
{       extern __shared__ byte s[];
		long length_N = length;
		T* temp=(T*)malloc(2*sizeof(T));
		int step = gridDim.x*blockDim.x;
		long start_P=start;//开始的位置
		long start_N =threadIdx.x+blockIdx.x*blockDim.x;
		for(long start=start_N;start<length_N;start=start+step)
		   {  if((char)*(info+start+start_P)=='\n')
		        {
		           len(info+start+start_P,temp);
		           if((int)temp[0]>(int)s[threadIdx.x*2]){
		        	   s[threadIdx.x*2]=temp[0];
		        	   if((int)temp[0]<1)
		        	     {T* temp_s=(T*)malloc(temp[1]*sizeof(T)+1+20);
						  memcpy(temp_s,info+start+start_P-(int)temp[1]*sizeof(T)-20,(int)temp[1]*sizeof(T)+1+20);
						  printf("s0:%c,s1:%d,s2:%d,小于:=%s \n",(char)(*(info+start+start_P-1)),(int)temp[0],(int)temp[1],temp_s);
						  printf("------------------------ \n");
						  delete temp_s;
		        	     }
		           }

		           if(threadIdx.x==27 and start==start_N){
		        	   T* temp_s=(T*)malloc(temp[1]*sizeof(T));
		        	   						  memcpy(temp_s,info+start+start_P-(int)temp[1]*sizeof(T),(int)temp[1]*sizeof(T));
		        	   						  printf("s0:%c,s1:%d,s2:%d,:=%s \n",(char)(*(info+start+start_P-1)),(int)temp[0],(int)temp[1],temp_s);
		        	   						  printf("------------------------ \n");
		        	   						  delete temp_s;
		           		        	   }

		           if((int)temp[1]>(int)s[threadIdx.x*2+1]){
		        	   s[threadIdx.x*2+1]=temp[1];
		           }

		           if((int)temp[0]>6)
		         				 {
		         				  T* temp_s=(T*)malloc(temp[0]*sizeof(T)+1);
		         				  memcpy(temp_s,info+start+start_P-temp[0]*sizeof(T),temp[0]*sizeof(T));
		         				  printf("大于6：=thread:=%d,%d,%s \n",(int)threadIdx.x,(int)temp[0],temp_s);
		         				  printf("--------------- \n");
		         				  delete temp_s;
		         				 }
		         			   temp[0]=0;
		         			   temp[1]=0;
		        }
		    }
		   delete temp;
		//同步
		__syncthreads();
		if(threadIdx.x==0)
		memcpy(dum+2*blockIdx.x*blockDim.x*sizeof(T),s,2*blockDim.x*sizeof(T));
		__syncthreads();
}

template __host__ __device__ void len<ubyte>(const char*,ubyte *);
template __host__ __device__ void len<byte>(const char*,byte *);
template __global__ void split_global<ubyte>(ubyte*, char*,long,long);
template __global__ void split_global<byte>(byte*, char*,long,long);

template __host__ __device__ void len<int>(const char*,int *);
template __global__ void split_global<int>(int*, char*,long,long);


