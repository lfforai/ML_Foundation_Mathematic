#include "hip/hip_runtime.h"
/*
 * cudalib.cpp
 *
 *  Created on: 2018年5月17日
 *      Author: root
 */

#include <src/cudalib.h>
#include <stdio.h>
#include "hip/hip_runtime.h"
#include "nccl.h"
#include <string>
#include "src/cudalib.h"

using namespace std;

namespace gpu {
__global__ void split_global(Matrix A, Matrix B, Matrix C)
{
		float Cvalue = 0;
		int row = blockIdx.y * blockDim.y + threadIdx.y;
		int col = blockIdx.x * blockDim.x + threadIdx.x;
		for (int e = 0; e < A.width; ++e)
			Cvalue += A.elements[row * A.width + e]
					* B.elements[e * B.width + col];
		C.elements[row * C.width + col] = Cvalue;
}

} /* namespace gpu */
