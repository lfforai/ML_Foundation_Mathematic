#include "hip/hip_runtime.h"
/*
 * cudalib.cpp
 *
 *  Created on: 2018年5月17日
 *      Author: root
 */

//#include <src/cudalib.h>
//#include <stdio.h>
//#include "hip/hip_runtime.h"
//#include "nccl.h"
//#include <string>
//#include "src/cudalib.h"
#include ""
#include "hip/hip_runtime.h"
#include "nccl.h"
#include "file_input.h"
#include <src/cudalib.h>
#include <hip/hip_fp16.h>

#include <string>
using namespace std;

//统计最大祖先个数、最大祖先长度和最大单条长度
template <class T>
__host__ __device__ void len(const char*info,T*result)
{
  int i=0;//index
  char frist_mark=(byte)0;
  while(*(info+i)!='@'){
	  if((*(info+i)=='.' or *(info+i)=='_')){
		  result[0]=(T)((int)result[0]+(int)1);
	  }

	  if(frist_mark==(byte)1)//if frist '.'
	  {result[1]=(T)((int)result[1]+(int)1);}

	  if(frist_mark==(byte)0 and *(info+i)=='.')//if frist '.'
	  { frist_mark=(byte)1;
	  }
	  i=i-1;
  }
  result[2]=(T)(abs(i)-2);
}

template <class T>
__global__ void split_global(T* dum, char* info,long start,long length,int dimblock)
{       extern __shared__ byte s[];
        if (threadIdx.x==0){
           memset(s,(byte)0,3*dimblock*sizeof(T));
        }
    	__syncthreads();
//		T* temp=(T*)malloc(2*sizeof(T));
		T temp[3];
		long length_N = length;
		int step = gridDim.x*blockDim.x;
		const long start_P=start;//开始的位置
		long start_N =threadIdx.x+blockIdx.x*blockDim.x;
		for(long start=start_N;start<length_N;start=start+step)
		   {
			  if((char)*(info+start+start_P)=='\n')
		        {  temp[0]=0;
				   temp[1]=0;
		           len(info+start+start_P,temp);
		           if((int)temp[0]>(int)s[threadIdx.x*3]){
		        	   s[threadIdx.x*3]=temp[0];
		           }

		           if((int)temp[1]>(int)s[threadIdx.x*3+2]){
		        	   s[threadIdx.x*3+1]=temp[1];
		           }

		           if((int)temp[2]>(int)s[threadIdx.x*3+2]){
		        	   s[threadIdx.x*3+2]=temp[2];
		           }

		        }
		    }
		delete temp;

		//同步
		__syncthreads();
		if(threadIdx.x==0)
		memcpy(dum+3*blockIdx.x*blockDim.x*sizeof(T),s,3*blockDim.x*sizeof(T));
		__syncthreads();
}

//切割出所有祖先，为放入hash表用
//dimGrid_N, dimBlock_N,0,s[i]>>>(d_result[i]+h_len_result[deviceCount-2]*max_an_len*max_an_num,d_info[i],max_an_len,max_an_num,(deviceCount-1)*sub_length,sub_length+yu,dimBlock_N
template <class T>
__global__ void scut2ancestors(char* des,int max_an_len,int max_an_num,char* info,long start,long length,long* mark,int dimblock)
{       //为每个thread分配空间记录当前分解记录的各"_"和“.”的位置
	    extern __shared__ byte s[];
		long length_N = length;
		int step = gridDim.x*blockDim.x;
		const long start_P=start;//开始的位置
		long start_N =threadIdx.x+blockIdx.x*blockDim.x;
		if(threadIdx.x==0){
			*mark=0;
			memset(s,(byte)0,max_an_num*dimblock*sizeof(T));
		}
		__syncthreads();

		for(long start=start_N;start<length_N;start=start+step)
		   {
			  if((char)*(info+start+start_P)=='\n')
		        { //1、-----------------------------------------------------
				  int dian_i=0;//第一个"."前面的最大祖先的根位置
				  int last_i=0;//开始的"@"

				  int first_dian_mark=0;//是否第一个"."
				  int i=0;
				  while(*(info+start+start_P+i)!='@')
		          {if(*(info+start+start_P+i)=='.' and first_dian_mark==0)
		        	  {dian_i=i;
		        	   first_dian_mark=1;
		        	  }
		           i=i-1;
		          }
				  last_i=i;
				  dian_i=dian_i-1;//最大祖先的开始位置info+start+start_P+dian_i
				  last_i=last_i+1;//最大祖先的结束位置info+start+start_P+last_i
				  //----------------------------------------------------

				  //2、记录各级祖先节点的位置----"."和“—”
				  int last_i_N=last_i;//保留祖先开始位置记录
				  int an_num=0;//祖先数目
				  while(last_i<=dian_i){
					  if(*(info+start+start_P+last_i)=='.' or *(info+start+start_P+last_i)=='_'){
						s[threadIdx.x*max_an_num+an_num]=(T)last_i;
						an_num=1+an_num;
					  }
					  last_i=last_i+1;
				  }
				  //最后一个祖先节点位置
				  s[threadIdx.x*max_an_num+an_num+1]=(T)last_i;

				  //3、依次输出各祖先节点
				  an_num=0;
				  long position=0;
				  while(s[threadIdx.x*max_an_num+an_num]!=0){
					  position=(long)atomicAdd((int *)mark,(int)1);
					  memcpy(des+position*max_an_len,info+start+start_P+dian_i,s[threadIdx.x*max_an_num+an_num]-last_i_N);
					  an_num=an_num+1;
				  }
				  if(threadIdx.x==0){
							memset(s,(byte)0,max_an_num*dimblock*sizeof(T));
						}
				  __syncthreads();
		        }
		    }

		//同步
		__syncthreads();
}

template __host__ __device__ void len<ubyte>(const char*,ubyte *);
template __host__ __device__ void len<byte>(const char*,byte *);
template __global__ void split_global<ubyte>(ubyte*, char*,long,long,int);
template __global__ void split_global<byte>(byte*, char*,long,long,int);

template __host__ __device__ void len<int>(const char*,int *);
template __global__ void split_global<int>(int*, char*,long,long,int);

template __global__ void scut2ancestors<byte>(char*,int ,int ,char*,long,long,long*,int);
template __global__ void scut2ancestors<ubyte>(char*,int ,int ,char*,long,long,long*,int);
template __global__ void scut2ancestors<int>(char*,int ,int ,char*,long,long,long*,int);
