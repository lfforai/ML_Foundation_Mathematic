#include "hip/hip_runtime.h"
/*
 * cudalib.cpp
 *
 *  Created on: 2018年5月17日
 *      Author: root
 */

//#include <src/cudalib.h>
//#include <stdio.h>
//#include "hip/hip_runtime.h"
//#include "nccl.h"
//#include <string>
//#include "src/cudalib.h"
#include ""
#include "hip/hip_runtime.h"
#include "nccl.h"
#include "file_input.h"
#include <src/cudalib.h>
#include <hip/hip_fp16.h>

#include <string>
using namespace std;

template <class T>
__host__ __device__ void len(const char*info,T*result)
{
  int i=0;//index
  char frist_mark=(byte)0;
  while(*(info+i)!='@'){
	  if((*(info+i)=='.' or *(info+i)=='_')){
		  result[0]=(T)((int)result[0]+(int)1);
	  }

	  if(frist_mark==(byte)1)//if frist '.'
	  {result[1]=(T)((int)result[1]+(int)1);}

	  if(frist_mark==(byte)0 and *(info+i)=='.')//if frist '.'
	  { frist_mark=(byte)1;
	  }
	  i=i-1;
  }
  result[2]=(T)(abs(i)-2);
}

template <class T>
__global__ void split_global(T* dum, char* info,long start,long length,int dimblock)
{       extern __shared__ byte s[];
        if (threadIdx.x==0){
           memset(s,(byte)0,3*dimblock*sizeof(T));
        }
    	__syncthreads();
//		T* temp=(T*)malloc(2*sizeof(T));
		T temp[3];
		long length_N = length;
		int step = gridDim.x*blockDim.x;
		const long start_P=start;//开始的位置
		long start_N =threadIdx.x+blockIdx.x*blockDim.x;
		for(long start=start_N;start<length_N;start=start+step)
		   {
			  if((char)*(info+start+start_P)=='\n')
		        {  temp[0]=0;
				   temp[1]=0;
		           len(info+start+start_P,temp);
		           if((int)temp[0]>(int)s[threadIdx.x*3]){
		        	   s[threadIdx.x*3]=temp[0];
//		        	   if((int)temp[0]<1)
//		        	     {T* temp_s=(T*)malloc((temp[1]+21)*sizeof(T));
//						  memcpy(temp_s,info+(start+start_P-(int)temp[1]-20)*sizeof(T),((int)temp[1]+21)*sizeof(T));
//						  printf("s0:%c,s1:%d,s2:%d,小于零:=%s \n",(char)(*(info+start+start_P-1)),(int)temp[0],(int)temp[1],temp_s);
//						  printf("------------------------ \n");
//						  delete temp_s;
//		        	     }
		           }

//		           if(threadIdx.x<3 and start==start_N){
//		        	   T* temp_s=(T*)malloc(temp[1]*sizeof(T));
//		        	   						  memcpy(temp_s,info+(start+start_P-(int)temp[1])*sizeof(T),(int)temp[1]*sizeof(T));
//		        	   						  printf("123：===index:%d,s0:%c,s1:%d,s2:%d,:=%s \n",threadIdx.x,(char)(*(info+start+start_P-1)),(int)temp[0],(int)temp[1],temp_s);
//		        	   						  printf("------------------------ \n");
//		        	   						  delete temp_s;
//		           		        	   }

		           if((int)temp[1]>(int)s[threadIdx.x*3+2]){
		        	   s[threadIdx.x*3+1]=temp[1];
		           }

		           if((int)temp[2]>(int)s[threadIdx.x*3+2]){
		        	   s[threadIdx.x*3+2]=temp[2];
		           }
//		           if((int)temp[0]>6)
//		         				 {printf("-------大于6开始-------- \n");
//		         				  T* temp_s=(T*)malloc((temp[1]+1)*sizeof(T));
//		         				  memcpy(temp_s,info+(start+start_P-temp[1])*sizeof(T),temp[1]*sizeof(T));
//		         				  printf("大于6：thread:=%d,最后一个:%c,temp0:=%d,temp1:=%d,内容:=%s \n",(int)threadIdx.x,(char)(*(info+start+start_P-1)),(int)temp[0],(int)temp[1],temp_s);
//		         				  printf("-------大于6结束-------- \n");
//		         				  delete temp_s;
//		         				 }
		        }
		    }
		delete temp;

		//同步
		__syncthreads();
		if(threadIdx.x==0)
		memcpy(dum+3*blockIdx.x*blockDim.x*sizeof(T),s,3*blockDim.x*sizeof(T));
		__syncthreads();
}

template __host__ __device__ void len<ubyte>(const char*,ubyte *);
template __host__ __device__ void len<byte>(const char*,byte *);
template __global__ void split_global<ubyte>(ubyte*, char*,long,long,int);
template __global__ void split_global<byte>(byte*, char*,long,long,int);

template __host__ __device__ void len<int>(const char*,int *);
template __global__ void split_global<int>(int*, char*,long,long,int);


