#include "hip/hip_runtime.h"
/*
 * cudalib.cpp
 *
 *  Created on: 2018年5月17日
 *      Author: root
 */

#include <src/cudalib.h>
#include <stdio.h>
#include "hip/hip_runtime.h"
#include "nccl.h"
#include <string>
#include "src/cudalib.h"

using namespace std;

__global__ void split_global(char* dum, char* info,long length)
{       extern __shared__ char s[];
		long length_N = length;
		int step = gridDim.x*blockDim.x;
		long start_N =threadIdx.x+blockIdx.x*blockDim.x;
		if (start_N==0){
			printf("d:=%d \n",gridDim.x);
		}
		for(long start=start_N;start<length_N;start=+step)
		   {
		   }
}

