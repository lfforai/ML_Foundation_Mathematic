#include "hip/hip_runtime.h"

/**
 * 作用：求所有key的最大祖先个数和最大key长度
 * 作者：罗峰
 */
#include "nccl.h"
#include ""
#include "hip/hip_runtime.h"
#include <src/cudalib.h>
#include <src/split.h>
#define CUDACHECK(cmd) do {                         \
  hipError_t e = cmd;                              \
  if( e != hipSuccess ) {                          \
    printf("Failed: Cuda error %s:%d '%s'\n",             \
        __FILE__,__LINE__,hipGetErrorString(e));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


#define NCCLCHECK(cmd) do {                         \
  ncclResult_t r = cmd;                             \
  if (r!= ncclSuccess) {                            \
    printf("Failed, NCCL error %s:%d '%s'\n",             \
        __FILE__,__LINE__,ncclGetErrorString(r));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)

template<typename T>
split<T>::split() {
	// TODO Auto-generated constructor stub
}

template<typename T>
split<T>::~split() {
	// TODO Auto-generated destructor stub
}

//返回所有key值中分解出的最多的一个祖先个数
//例如：MDNY.SZWQ.11MD#1.MlyGnCptLo,祖先个数为3
//返回keys列表中单个key最大祖先个数、最大祖先字符串长度，本身字符串最大长度
template<typename T>
T* split<T>::max_ancestors_num(file_input::info* info_of_key,int GPU_num){
	int dimGrid_N=224;
	int dimBlock_N=256;
	int result_n=3;//需要输出几种结果：目前为3

    T* max=(T*)malloc(result_n*sizeof(T));//max[0]:祖先最大数，max[1]:key的最大长度
    max[0]=(T)0;
    max[1]=(T)0;
    max[2]=(T)0;

    char* keys_data=info_of_key->data;
//  long row_num=info_of_key->total_row;
    long buffer_size=info_of_key->total_size;//字节数
    int deviceCount=GPU_num;

    if (GPU_num==-1)
	   {CUDACHECK(hipGetDeviceCount(&deviceCount));}

    //#记录每个key的最大祖先个数和最大长度的数组
    T** h_num=(T **)malloc(deviceCount*sizeof(T*));
    T** d_num=(T **)malloc(deviceCount*sizeof(T*));
    char** d_info=(char **)malloc(deviceCount*sizeof(char*));

    long yu= buffer_size%deviceCount;
    long sub_length=buffer_size/deviceCount;

    for(int i=0;i<2;i++)
    {hipOccupancyMaxPotentialBlockSize(
		&dimGrid_N,
		&dimBlock_N,
		(void*)split_global<T>,
		result_n*dimBlock_N*sizeof(T),
		2048);
        printf("第%d次：dimGrid_N=:%d,dimBlock_N:=%d \n",i,dimGrid_N,dimBlock_N);
    }

    //此处假设所有的祖先不超过255个，所以采用char
    for(int i=0;i<deviceCount;i++){
    	CUDACHECK(hipSetDevice(i));
    	h_num[i]=(T *)malloc(result_n*dimGrid_N*dimBlock_N*sizeof(T));
        CUDACHECK(hipMalloc(d_num+i,result_n*dimGrid_N*dimBlock_N*sizeof(T)));
        CUDACHECK(hipMalloc(d_info+i,buffer_size* sizeof(char)));
        if (i==0){//gpu较多情况下使用nccl，在gpu较少情况下可以不使用nccl，这里统一使用nccl无论gpu个数
           CUDACHECK(hipMemcpy(d_info[i],keys_data,buffer_size*sizeof(char), hipMemcpyHostToDevice));
        }
    }

//把 d_info[0]通过nccl的boadcast到d_info[i]上去--------------------开始
     hipStream_t* s = (hipStream_t*)malloc(sizeof(hipStream_t)*deviceCount);
     ncclComm_t* comms=(ncclComm_t*)malloc(deviceCount*sizeof(ncclComm_t));
     //managing deviceCount devices
     int* devs=(int *)malloc(deviceCount*sizeof(int));
     for(int i=0;i<deviceCount;i++){
    	  CUDACHECK(hipSetDevice(i));
    	  devs[i]=i;
    	  CUDACHECK(hipStreamCreate(s+i));
     }

     //initializing NCCL
     NCCLCHECK(ncclCommInitAll(comms,deviceCount, devs));
     //calling NCCL communication API. Group API is required when using
     //multiple devices per thread
     NCCLCHECK(ncclGroupStart());
     for (int i = 0; i < deviceCount; ++i)
   	     NCCLCHECK(ncclBcast((void*)d_info[i],buffer_size,ncclChar,0,comms[i], s[i]));
     NCCLCHECK(ncclGroupEnd());
     //synchronizing on CUDA streams to wait for completion of NCCL operation
     for (int i = 0; i < deviceCount; ++i) {
       CUDACHECK(hipSetDevice(i));
       CUDACHECK(hipStreamSynchronize(s[i]));
     }
     //finalizing NCCL
      for(int i = 0; i <deviceCount; ++i)
          {ncclCommDestroy(comms[i]);
           CUDACHECK(hipStreamDestroy(s[i]));}
//把 d_info[0]通过nccl的boadcast到d_info[i]上去--------------------开始

      s = (hipStream_t*)malloc(sizeof(hipStream_t)*deviceCount);
      for(int i=0;i<deviceCount;i++){
     	  CUDACHECK(hipSetDevice(i));
     	  CUDACHECK(hipStreamCreate(s+i));
      }

      for(int i=0;i<deviceCount;i++)
      {   CUDACHECK(hipSetDevice(i));
          if (i!=deviceCount-1)
          {
    	   split_global<T><<<dimGrid_N, dimBlock_N,result_n*dimBlock_N*sizeof(T),s[i]>>>(d_num[i],d_info[i],i*sub_length,sub_length,dimBlock_N);
    	   //2*dimBlock_N*sizeof(T)
          }
          else
          {
    	   split_global<T><<<dimGrid_N, dimBlock_N,result_n*dimBlock_N*sizeof(T),s[i]>>>(d_num[i],d_info[i],(deviceCount-1)*sub_length,sub_length+yu,dimBlock_N);
          }
      }

      for (int i = 0;i < deviceCount;i++)
        {
            CUDACHECK(hipSetDevice(i));
            CUDACHECK(hipMemcpyAsync(h_num[i],d_num[i],result_n*dimGrid_N*dimBlock_N*sizeof(T),hipMemcpyDeviceToHost,s[i]));
        }


      for (int i = 0;i < deviceCount;i++)
      {
          CUDACHECK(hipSetDevice(i));
          CUDACHECK(hipStreamSynchronize(s[i]));
      }

      for(int i = 0; i <deviceCount; ++i)
           {CUDACHECK(hipStreamDestroy(s[i]));}

      for (int i = 0; i < deviceCount;i++)
      {  for (int j = 0; j < dimGrid_N*dimBlock_N;j++)
          {
        	 if((int)h_num[i][j*result_n]>(int)max[0]){
        		 max[0]=h_num[i][j*result_n];
//        		 printf("max[0]：=%d \n",max[0]);
        	 }

        	 if((int)h_num[i][j*result_n+1]>(int)max[1]){
        		 max[1]=h_num[i][j*result_n+1];
//        		 printf("max[1]：=%d \n",max[1]);
        	 }

        	 if((int)h_num[i][j*result_n+2]>(int)max[2]){
        		 max[2]=h_num[i][j*result_n+2];
//        		 printf("max[1]：=%d \n",max[1]);
        	 }
//        	     if(j<2 and (h_num[i][j*2]!=0 or h_num[i][j*2+1]!=0))
//        	     printf("max[0]=%d,max[1]=%d,i:=%d,j:=%d \n",(int)h_num[i][j*2],(int)h_num[i][j*2+1],i,j);
          }
      }

      printf("%d,%d,%d \n",max[0],max[1],max[2]);
      return max;
};

template class split<byte>;
template class split<ubyte>;
template class split<int>;

