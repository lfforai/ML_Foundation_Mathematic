#include "hip/hip_runtime.h"
/*
 * split.h
 *
 *  Created on: 2018年5月17日
 *       Author: 罗锋
 *  基于多GPU集群的族谱分解：
 *  例如：MDNY.SZWQ.11MD#1.MlyGnCptLo拆分为
 *       grand-grand-father：MDNY
 *       grand-father：MDNY.SZWQ
 *       father：MDNY.SZWQ.11MD#1
 */
#include "hip/hip_runtime.h"
//include的输入是有顺序
#include "nccl.h"
#include <src/split.h>
#include <string.h>
#include <stdlib.h>
#include <stdio.h>
#include <src/cudalib.h>

//#include <src/cudalib.h>
using namespace std;

split::split() {
	// TODO Auto-generated constructor stub

}

split::~split() {
	// TODO Auto-generated destructor stub
}

//返回所有key值中分解出的最多的一个祖先个数
//例如：MDNY.SZWQ.11MD#1.MlyGnCptLo,祖先个数为3
//返回keys列表中单个key最大祖先个数和最大字符串长度
int* split::max_ancestors_num(file_input::info* info_of_key,int GPU_num){
	int dimGrid_N=224;
	int dimBlock_N=256;

	int* max=(int*)malloc(2*sizeof(int));//max[0]:祖先最大数，max[1]:key的最大长度
    char* keys_data=info_of_key->data;
    long row_num=info_of_key->total_row;
    long buffer_size=info_of_key->total_size;//字节数
    printf("sizeof:=%ld",buffer_size);
    int  deviceCount=GPU_num;

    if (GPU_num==-1)
	   {CUDACHECK(hipGetDeviceCount(&deviceCount));}

//    #记录每个key的最大祖先个数和最大长度的数组
    char** h_num=(char **)malloc(deviceCount*sizeof(char*));
    //每个gpu一个数组记录最大值
    char** d_num=(char **)malloc(deviceCount*sizeof(char*));
    char** d_info=(char **)malloc(deviceCount*sizeof(char*));

    long yu= row_num%deviceCount;
    long sub_length= row_num/deviceCount;

    //此处假设所有的祖先不超过255个，所以采用char
    for(int i=0;i<deviceCount;i++){
    	CUDACHECK(hipSetDevice(i));
    	//每个gpu一个数组记录：每个线程获取的部分key值中最大的祖先数
        CUDACHECK(hipMalloc(d_num + i,dimGrid_N*dimBlock_N*sizeof(char)));
        h_num[i]=(char *)malloc(dimGrid_N*dimBlock_N*sizeof(char*));
        CUDACHECK(hipMalloc(d_info + i,buffer_size* sizeof(char)));
        if (i==0){//gpu较多情况下使用nccl，在gpu较少情况下可以不使用nccl，这里统一使用nccl无论gpu个数
           CUDACHECK(hipMemcpy(d_info[i],keys_data,buffer_size*sizeof(char), hipMemcpyHostToDevice));
        }
    }
//把 d_info[0]通过nccl的boadcast到d_info[i]上去--------------------开始
     ncclComm_t* comms=(ncclComm_t*)malloc(deviceCount*sizeof(ncclComm_t));
     //managing deviceCount devices
     int* devs=(int *)malloc(deviceCount*sizeof(int));
     for(int i=0;i<deviceCount;i++){
    	  devs[i]=i;
     }
     hipStream_t* s = (hipStream_t*)malloc(sizeof(hipStream_t)*deviceCount);

     //initializing NCCL
     NCCLCHECK(ncclCommInitAll(comms,deviceCount, devs));
     //calling NCCL communication API. Group API is required when using
     //multiple devices per thread

     NCCLCHECK(ncclGroupStart());
     for (int i = 0; i < deviceCount;i++)
     	  NCCLCHECK(ncclBcast((void*)d_info[i],(size_t)buffer_size,ncclChar,0,comms[i], s[i]));

     NCCLCHECK(ncclGroupEnd());

     //synchronizing on CUDA streams to wait for completion of NCCL operation
     for (int i = 0; i < deviceCount;i++) {
       CUDACHECK(hipSetDevice(i));
       CUDACHECK(hipStreamSynchronize(s[i]));
     }
     //finalizing NCCL
     for(int i = 0; i <deviceCount;i++)
          ncclCommDestroy(comms[i]);
//把 d_info[0]通过nccl的boadcast到d_info[i]上去--------------------开始
     printf("kaishi1 \n");
     for(int i=0;i<deviceCount;i++){
       if (i!=deviceCount-1)
          {CUDACHECK(hipSetDevice(i));
    	   split_global<<<dimGrid_N, dimBlock_N,dimBlock_N>>>(d_num[i],d_info[i]+i*sub_length,sub_length);}
       else
          {CUDACHECK(hipSetDevice(i));
    	   split_global<<<dimGrid_N, dimBlock_N,dimBlock_N>>>(d_num[i],d_info[i]+(deviceCount-1)*sub_length,sub_length+yu);}
    }
	   CUDACHECK(hipDeviceSynchronize());

   	return max;
};

