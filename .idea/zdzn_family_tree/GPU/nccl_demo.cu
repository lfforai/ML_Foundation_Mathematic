//demo作用：多gpu可以通过nccl构建gpu集群并行运算，nccl是nvidia开发的多gpu的信息交互工具类似MPI
//demo内容利用nccl中的例子中的=单thread对多device的模式，对reduce，allreduce,allgather,ReduceScatter进行了测试
//参考文档:https://docs.nvidia.com/deeplearning/sdk/nccl-developer-guide/index.html
//作者：罗锋
//demo归类：数据族谱gpu实现的子项目测试

#include <stdio.h>
#include "hip/hip_runtime.h"
#include "nccl.h"
#include <string>
using namespace std;

#define CUDACHECK(cmd) do {                         \
  hipError_t e = cmd;                              \
  if( e != hipSuccess ) {                          \
    printf("Failed: Cuda error %s:%d '%s'\n",             \
        __FILE__,__LINE__,hipGetErrorString(e));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)

#define NCCLCHECK(cmd) do {                         \
  ncclResult_t r = cmd;                             \
  if (r!= ncclSuccess) {                            \
    printf("Failed, NCCL error %s:%d '%s'\n",             \
        __FILE__,__LINE__,ncclGetErrorString(r));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)

int main(int argc, char* argv[])
{
  ncclComm_t comms[2];

  //managing 4 devices
  int nDev = 2;
  int size = 10;
  int devs[2] = { 0, 1};

  //allocating and initializing device buffers
  float** sendbuff = (float**)malloc(nDev * sizeof(float*));
  float** recvbuff = (float**)malloc(nDev * sizeof(float*));
  float** h_sendbuff_N=(float **)malloc(nDev* sizeof(float*));
  hipStream_t* s = (hipStream_t*)malloc(sizeof(hipStream_t)*nDev);

  for(int i = 0; i < nDev; ++i)
  {
    CUDACHECK(hipSetDevice(i));
    CUDACHECK(hipMalloc(sendbuff + i, size * sizeof(float)));
    CUDACHECK(hipMalloc(recvbuff + i, size * sizeof(float)));
    for(int j=0;j<size;j++){
       h_sendbuff_N[i]=(float *)malloc(size * sizeof(float));
       for(int j=0;j<size;j++)
          {if (i==0 and j<5)
    	      *(h_sendbuff_N[i]+j)=1;
           else if(i==1 and j>=5)
                  *(h_sendbuff_N[i]+j)=2;
                else
                  {*(h_sendbuff_N[i]+j)=-1;}
          }
    }
    CUDACHECK(hipMemcpy(sendbuff[i], h_sendbuff_N[i], size*sizeof(float), hipMemcpyHostToDevice));
   // CUDACHECK(cudaMemset(sendbuff[i], 1, size * sizeof(float)));
    CUDACHECK(hipMemset(recvbuff[i],0,size * sizeof(float)));
    CUDACHECK(hipStreamCreate(s+i));
  }

  //initializing NCCL
  NCCLCHECK(ncclCommInitAll(comms, nDev, devs));

  //calling NCCL communication API. Group API is required when using
  //multiple devices per thread
  NCCLCHECK(ncclGroupStart());

  size_t sendcount=(int)(size/2);
  for (int i = 0; i < nDev; ++i)
//     NCCLCHECK(ncclAllReduce((const void*)sendbuff[i], (void*)recvbuff[i], size, ncclFloat, ncclSum,
//         comms[i], s[i]));
	   NCCLCHECK(ncclReduce((const void*)sendbuff[i], (void*)recvbuff[i], size, ncclFloat, ncclSum,
	           1,comms[i], s[i]));
//	   NCCLCHECK(ncclAllGather((const void*)(sendbuff[i]+i*sendcount), (void*)recvbuff[i], sendcount, ncclFloat,
//	       comms[i], s[i]));
//	   NCCLCHECK(ncclReduceScatter((const void*)sendbuff[i], (void*)(recvbuff[i]+i*sendcount), sendcount, ncclFloat,ncclSum,
//		   comms[i], s[i]));
  NCCLCHECK(ncclGroupEnd());

  //synchronizing on CUDA streams to wait for completion of NCCL operation
  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(i));
    CUDACHECK(hipStreamSynchronize(s[i]));
  }

  //输出结果产看
  float ** h_recvbuff=(float **)malloc(nDev*sizeof(float*));
  for(int i = 0; i < nDev; ++i){
	  *(h_recvbuff+i)=(float *)malloc(size*sizeof(float));
  }

  for (int i = 0; i < nDev; ++i) {
      CUDACHECK(hipSetDevice(i));
      CUDACHECK(hipMemcpy(*(h_recvbuff+i),recvbuff[i],size*sizeof(float), hipMemcpyDeviceToHost));
      for (int j=0;j<size;j++)
          {printf("result:=%f \n",*(*(h_recvbuff+i)+j));}
      printf("i:=%d \n",i);
  }

  //free device buffers
  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(i));
    CUDACHECK(hipFree(sendbuff[i]));
    CUDACHECK(hipFree(recvbuff[i]));
  }

  //finalizing NCCL
  for(int i = 0; i < nDev; ++i)
      ncclCommDestroy(comms[i]);

  printf("Success \n");
}
