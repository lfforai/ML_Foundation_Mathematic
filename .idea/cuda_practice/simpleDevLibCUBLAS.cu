#备注：多gpu之间信息交互的nccl2实现了:allreduce Reduce Broadcast ReduceScatter AllGather

/* Includes, system */
#include <stdio.h>
#include "hip/hip_runtime.h"
#include "nccl.h"
using namespace std;

#define CUDACHECK(cmd) do {                         \
  hipError_t e = cmd;                              \
  if( e != hipSuccess ) {                          \
    printf("Failed: Cuda error %s:%d '%s'\n",             \
        __FILE__,__LINE__,hipGetErrorString(e));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
}while(0)

#define NCCLCHECK(cmd) do {                         \
  ncclResult_t r = cmd;                             \
  if (r!= ncclSuccess) {                            \
    printf("Failed, NCCL error %s:%d '%s'\n",             \
        __FILE__,__LINE__,ncclGetErrorString(r));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
}while(0)


int main(int argc, char* argv[])
{
  ncclComm_t comms[2];
  //managing 4 devices
  int nDev = 2;
  int size = 32*1024*1024;
  int devs[4] = {0, 1};
  //allocating and initializing device buffers
  float** sendbuff = (float**)malloc(nDev * sizeof(float*));
  float** recvbuff = (float**)malloc(nDev * sizeof(float*));

  float** h_recvbuff=(float**)malloc(nDev * sizeof(float*));//host
  float** h_sendbuff=(float**)malloc(nDev * sizeof(float*));//host

  for (int i = 0; i < nDev; ++i){
	  h_sendbuff[i]=(float*)malloc(size * sizeof(float));
	  for(int j=0;j<size;++j){
		  if(j<100)
		  {h_sendbuff[i][j]=1;}
		  else
		  {if(j>=100 && j<200)
			  h_sendbuff[i][j]=2;
		   else
			  h_sendbuff[i][j]=3;
		  }
	  }
  }

  hipStream_t* s =(hipStream_t*)malloc(sizeof(hipStream_t)*nDev);
  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(i));
    CUDACHECK(hipMalloc(sendbuff + i, size * sizeof(float)));
    CUDACHECK(hipMalloc(recvbuff + i, size * sizeof(float)));
    CUDACHECK(hipMemset(recvbuff[i], 0.0, size * sizeof(float)));
    h_recvbuff[i]=(float*)malloc(size * sizeof(float));
    CUDACHECK(hipStreamCreate(s+i));
  }

  //initializing NCCL
  NCCLCHECK(ncclCommInitAll(comms, nDev, devs));
  //calling NCCL communication API. Group API is required when using

  //multiple devices per thread
  for(int i = 0; i < nDev; ++i){
	 CUDACHECK(hipSetDevice(i));
	 CUDACHECK(hipMemcpyAsync(sendbuff[i],h_sendbuff[i],size*sizeof(float),hipMemcpyHostToDevice,s[i]));
  }

  NCCLCHECK(ncclGroupStart());
  for (int i = 0; i < nDev; ++i)
    { //1、把每个recvbuff[i]=sendbuff[1]+sendbuff[2]+.......sendbuff[nDev]
	  //NCCLCHECK(ncclAllReduce((const void*)sendbuff[i],(void*)sendbuff[i],
    		  //size, ncclFloat, ncclSum,comms[i], s[i]));

	  //2、第root个recvbuff[i]=sendbuff[1]+sendbuff[2]+.......sendbuff[nDev]
	  //NCCLCHECK(ncclReduce((const void*)sendbuff[i],(void*)recvbuff[i],size, ncclFloat, ncclSum,0,comms[i], s[i]));

	 //3、第root个sendbuff[i]的前count个数据，分发到所有其他每个recvbuff[i]上去
	 //recvbuff[1023]=1,而recvbuff[1024]=0
	 //NCCLCHECK(ncclBroadcast((const void*)sendbuff[i],(void*)recvbuff[i],1024,ncclFloat,0,comms[i], s[i]));

	 //4、在(void*)recvbuff[i]+100*i,recvbuff[i]+100*i+100范围内的reduce，以外的为0
	   //NCCLCHECK(ncclReduceScatter((const void*)sendbuff[i],(void*)recvbuff[i]+100*i,100,ncclFloat,ncclSum,comms[i],s[i]));
	 //5、
	  NCCLCHECK(ncclAllGather((const void*)sendbuff[i]+100*i,(void*)recvbuff[i],100,ncclFloat,comms[i],s[i]));
    }
  NCCLCHECK(ncclGroupEnd());

  //synchronizing on CUDA streams to wait for completion of NCCL operation
  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(i));
    CUDACHECK(hipStreamSynchronize(s[i]));
  }

  for(int i = 0; i < nDev; ++i){
	 CUDACHECK(hipSetDevice(i));
	 CUDACHECK(hipMemcpyAsync(h_recvbuff[i],recvbuff[i],size*sizeof(float),hipMemcpyDeviceToHost,s[i]));
	 CUDACHECK(hipMemcpyAsync(h_sendbuff[i],sendbuff[i],size*sizeof(float),hipMemcpyDeviceToHost,s[i]));
  }

  //free device buffers
  for (int i = 0; i < nDev; ++i) {
    CUDACHECK(hipSetDevice(i));
    CUDACHECK(hipFree(sendbuff[i]));
    CUDACHECK(hipFree(recvbuff[i]));
  }

  for (int i = 0; i < nDev; ++i) {
	  printf("%d,recvbuff=%f,sendbuff=%f \n",i,h_recvbuff[i][100*i+9],h_sendbuff[i][10]);
	  printf("out:=%d,recvbuff=%f,sendbuff=%f \n",i,h_recvbuff[i][100*i+110],h_sendbuff[i][10]);
  }

  //finalizing NCCL
  for(int i = 0; i < nDev; ++i)
      ncclCommDestroy(comms[i]);
  printf("Success \n");
  return 0;
}
